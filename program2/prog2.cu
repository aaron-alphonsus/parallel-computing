/* File:                                                                         
 *    prog2.cu
 *                                                                               
 * Purpose:                                                                      
 *    TODO A brute force solution to a circuit-satisfiability question parallelizing
 *    the algorithm using OpenMP. Compares time taken by parallel and serial 
 *    approaches.     
 *                                                                               
 * Input:                                                                        
 *    TODO none (pre-defined circuit in the form of an if statement)                            
 * Output:                                                                       
 *    TODO All combinations of inputs that satisfy the circuit.
 *                                                                               
 * Compile:
 *    nvcc -o prog2 prog2.cu
 *     OR 
 *    make
 * Usage:                                                                        
 *    ./prog2 (To profile: nvprof ./prog2)
 *                                                                               
 * Professor:                                                                    
 *    Dr. Christer Karlsson                                                      
 * Authors:                                                                      
 *    Aaron Alphonsus                                                            
 * Class:                                                                        
 *    CSC410 - Parallel Computing                                                
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
// #include <stdlib.h>
// #include <math.h>

__global__ void matvecMul(double *A, double *B, double *C, int n)
{
    double sum = 0;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for(int i = index; i < n; i += stride)
    {
        sum = 0; 
        for(int j = 0; j < n; j++)
            sum += A[i * n + j] * B[j];
        C[i] = sum;
    }
}

int main(int argc, char* argv[])
{ 
    // Size of vectors
    int n = 8192;
    double sumofsq = 0;
    unsigned long long int wrong = 0;

    // Device input vectors
    double *A;
    double *B;
    // Device output vectors
    double *C;

    // Size, in bytes, of 'A' "matrix"
    size_t mat_bytes = n * n * sizeof(double); 
    // Size, in bytes, of 'B' and 'C' vectors
    size_t vec_bytes = n * sizeof(double);

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&A, mat_bytes);
    hipMallocManaged(&B, vec_bytes);
    hipMallocManaged(&C, vec_bytes);

    // Initialize vectors on host
    for(int i = 0; i < n; i++) 
        for(int j = 0; j < n; j++)
            A[i * n + j] = j; 
    for(int i = 0; i < n; i++) 
        B[i] = i;

    // int padding = 3;
    // printf("Matrix A\n");
    // for(int i = 0; i < n; i++) 
    // {
    //     for(int j = 0; j < n; j++)
    //         printf("%*lld", padding, A[i * n + j]);
    //     printf("\n");
    // }
    // printf("Matrix B\n");
    // for(int i = 0; i < n; i++) 
    //     printf("%lld ", B[i]);
    // printf("\n");

    // No. of threads in each thread block and no. of thread blocks in grid
    int blockSize = 256;
    int gridSize = (int)ceil((float)n/blockSize);
    // Execute the kernel
    matvecMul<<<gridSize, blockSize>>>(A, B, C, n);
    // matvecMul<<<1, 128>>>(A, B, C, n);

    // matvecMul_serial(A, B, C, n);

    // Wait for the GPU to finish
    hipDeviceSynchronize();
  
    for(int i = 0; i < n; i++)
        sumofsq += i*i;
    
    printf("Matrix C\n");
    for(int i = 0; i < n; i++) 
        printf("%0.1lf ", C[i]);
    printf("\n\n");
 
    // Check each C element with sumofsq
    for(int i = 0; i < n; i++) 
        if(sumofsq != C[i])
            wrong++;
    printf("Number of positions incorrect = %lld\n", wrong); 

    printf("n, sumofsq = %d, %0.1lf\n", n, sumofsq);
    printf("<<<gridSize, blockSize>>> = <<<%d, %d>>>\n\n", gridSize, blockSize);

    // Release Unified Memory 
    hipFree(A);
    hipFree(B);
    hipFree(C);

    return 0;
}
